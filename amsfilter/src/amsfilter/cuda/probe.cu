#include "probe.hpp"

#include <dtl/dtl.hpp>

#include <dtl/filter/blocked_bloomfilter/blocked_bloomfilter_config.hpp>
#include <dtl/filter/blocked_bloomfilter/blocked_bloomfilter_logic.hpp>

#include <amsfilter/internal/blocked_bloomfilter_template.hpp>
#include <amsfilter/internal/blocked_bloomfilter_resolve.hpp>

#include "internal/probe_impl.cuh"

namespace amsfilter {
namespace cuda {
//===----------------------------------------------------------------------===//
struct Probe::impl {

  /// The actual probe instance.
  internal::probe_impl instance;

  impl(const Config& config, const std::size_t desired_length)
      : instance(config, desired_length) {}
  ~impl() = default;
  impl(impl&&) = default;
  impl(const impl&) = delete;
  impl& operator=(impl&&) = delete;
  impl& operator=(const impl&) = delete;

};
//===----------------------------------------------------------------------===//
Probe::Probe(const AmsFilter& filter)
    : pimpl_{std::make_unique<impl>(filter.get_config(),
          filter.get_desired_length())} {}
Probe::Probe(Probe&&) noexcept = default;
Probe& Probe::operator=(Probe&& other) noexcept = default;
Probe::~Probe() = default;

void
Probe::operator()(
    const amsfilter::internal::word_t* __restrict filter_data,
    const amsfilter::internal::key_t* __restrict keys, u32 key_cnt,
    amsfilter::internal::word_t* __restrict result_bitmap,
    const hipStream_t& hip_stream) {
  pimpl_->instance.contains(filter_data, keys, key_cnt, result_bitmap,
      hip_stream);
}
//===----------------------------------------------------------------------===//
} // namespace cuda
} // namespace amsfilter
